#define N 512
#define NUM_BLOCKS 16
#define NUM_THREADS 48
//Do not change above three lines.

//Submission should be named as  <RollNumber>_Prog.cu
//Upload just this cu file and nothing else. If you upload it as a zip, it will not be evaluated.

/*Remember the following guidelines to avoid losing marks
This exercise is quite simple. The only tricky part is that total number of threads (NUM_BLOCKS*NUM_THREADS) may be different (higher or lower) from N.

Index of an array should not exceed the array size.

No output array-element should be computed more than once

No marks will be given if the program does not compile or run (TAs will not debug your program at all)

Do not change the name of any variable that we have introduced.
*/



#include <hip/hip_runtime.h>
#include <stdio.h>



//TODO: WRITE GPU KERNEL. It should not be called repeatedly from the host, but just once. Each time it is called, it may process more than array-element or not process any array-element at all.
__global__ void add(int *a, int *b, int *c,int length,int t){
        int index=0;
        for(int i=0;i<=t;i++){
                index= gridDim.x*blockDim.x*i;
                index+= threadIdx.x+blockIdx.x * blockDim.x;
                if(index < length)
                c[index]=a[index]+b[index];
        }
}

int main (int argc, char **argv) {
  int A[N][N], B[N][N], C[N][N];
  int *d_A, *d_B, *d_C; // These are the copies of A, B and C on the GPU
  int *h_C;       // This is a host copy of the output of B from the GPU
  int i, j;


  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      A[i][j] = i+j;
      B[i][j]= 2*j-1;
    }
  }

  // sequential implementation of main computation
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      C[i][j] = A[i][j]+B[i][j];
    }
  }

  // TODO: ALLOCATE MEMORY FOR GPU COPIES OF d_A, d_B and d_C
        int size= N*N*sizeof(int);
        hipMalloc((void **)&d_A, size);
        hipMalloc((void **)&d_B, size);
        hipMalloc((void **)&d_C, size);
  h_C=(int*)malloc(size);

  // TODO: COPY A TO d_A
        hipMemcpy(d_A,*A,size,hipMemcpyHostToDevice);

  // TODO: COPY B TO d_B
        hipMemcpy(d_B,*B,size,hipMemcpyHostToDevice);

  int length=N*N;
  int t=length/(NUM_BLOCKS*NUM_THREADS);

  // TODO: CREATE BLOCKS with THREADS AND INVOKE GPU KERNEL
  add<<<NUM_THREADS,NUM_BLOCKS>>>(d_A,d_B,d_C,length,t);

   //Use NUM_BLOCKS blocks, each with NUM_THREADS  threads

  // TODO: COPY d_C BACK FROM GPU to CPU in variable h_C
  hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
  int temp;
  int error=0;
  // TODO: Verify result is correct by comparing
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
    //TODO: compare each element of h_C and C by subtracting them
        //print only those elements for which the above subtraction is non-zero
         temp=h_C[i*N+j]-C[i][j];
        if(temp!=0){
        error++;
        printf("i=%d, j=%d, difference=%d\n",i,j,temp);
      }
    }
   }
  if(error!=0)printf("Total errors=%d\n",error);
    //IF even one element of h_C and C differ, report an error.
    //Otherwise, there is no error.
    //If your program is correct, no error should occur.
}